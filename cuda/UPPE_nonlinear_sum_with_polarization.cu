
#include <hip/hip_runtime.h>
#define MAX_NUM_MODES 32 // the maximum number of modes for this cuda = sqrt(MaxThreadsPerBlock)
                         //                                           = sqrt(1024) for our Titan XP GPU

__global__ void UPPE_nonlinear_sum_with_polarization(double2* Kerr, double2* Ra, double2* Rb, double2* Ra_sponRS, double2* Rb_sponRS,
                                                     const double2* At, const double2* At_noise,
                                                     const double* SK,  const unsigned char* SK_nonzero_midx1234s,  const unsigned int* SK_beginning_nonzero,  const unsigned int* SK_ending_nonzero,
                                                     const double* SRa, const unsigned char* SRa_nonzero_midx1234s, const unsigned int* SRa_beginning_nonzero, const unsigned int* SRa_ending_nonzero,
                                                     const double* SRb, const unsigned char* SRb_nonzero_midx1234s, const unsigned int* SRb_beginning_nonzero, const unsigned int* SRb_ending_nonzero,
                                                     const bool include_Raman, const bool include_anisoRaman,
                                                     const unsigned int N, const unsigned int M,
                                                     const unsigned int NUM_MODES,
                                                     const unsigned int NUM_OPERATIONS) {
    const unsigned int midx1 = threadIdx.x / NUM_MODES;
    const unsigned int midx2 = threadIdx.x - midx1*NUM_MODES;

    const unsigned int NMIdx = blockIdx.x / NUM_OPERATIONS;
    const unsigned int OPERATIONIdx = blockIdx.x - NMIdx*NUM_OPERATIONS;

    const unsigned int Midx = NMIdx / N;
    const unsigned int Nidx = NMIdx - Midx*N;

    const unsigned int NM = N*M;
    const unsigned int NMMODES = NM*NUM_MODES;

    // Preload At to improve the performance (avoiding accessing the global memory too many times)
    __shared__ double2 this_At[MAX_NUM_MODES], this_At_noise[MAX_NUM_MODES];
    switch (OPERATIONIdx) {
        case 0: // For Kerr interactions, noise photon is included directly for accurately computing noise-seeded processes
            if (midx1 == 0) {
                this_At[midx2].x = At[Nidx+Midx*N+midx2*NM].x + At_noise[Nidx+Midx*N+midx2*NM].x;
                this_At[midx2].y = At[Nidx+Midx*N+midx2*NM].y + At_noise[Nidx+Midx*N+midx2*NM].y;
            }
            break;
        case 1:
        case 2:
            if (midx1 == 0) this_At[midx2] = At[Nidx+Midx*N+midx2*NM];
            break;
        case 3:
        case 4:
            if (midx1 == 0) {
                this_At[midx2] = At[Nidx+Midx*N+midx2*NM];
                this_At_noise[midx2] = At_noise[Nidx+Midx*N+midx2*NM];
            }
            break;
    }
    __syncthreads();

    const unsigned int this_SK_beginning_nonzero = SK_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SK_ending_nonzero = SK_ending_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRa_beginning_nonzero = SRa_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRa_ending_nonzero = SRa_ending_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRb_beginning_nonzero = SRb_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRb_ending_nonzero = SRb_ending_nonzero[midx2+midx1*NUM_MODES];

    unsigned int midx3, midx4;
    double c, d, e, f;
    switch (OPERATIONIdx) {
        case 0: // compute the Kerr term
            if (this_SK_beginning_nonzero > 0) {
                double a, b, pcdef, ncdef;
                a = this_At[midx2].x;
                b = this_At[midx2].y;

                double2 this_Kerr;
                this_Kerr.x = 0; this_Kerr.y = 0; // initialized
                for (int i = this_SK_beginning_nonzero-1; i < this_SK_ending_nonzero-1; i++) {
                    midx3 = SK_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SK_nonzero_midx1234s[3+i*4]-1;
            
                    c = this_At[midx3].x;
                    d = this_At[midx3].y;
                    e = this_At[midx4].x;
                    f = this_At[midx4].y;
            
                    pcdef = SK[i]*(c*e+d*f);
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f --> ncdef=0
                            this_Kerr.x += a*pcdef;
                            this_Kerr.y += b*pcdef;
                        } else {
                            ncdef = SK[i]*(c*f-d*e);
                            this_Kerr.x += a*pcdef+b*ncdef;
                            this_Kerr.y += b*pcdef-a*ncdef;
                        }
                    } else { // (d*e-c*f) + (c <--> e, d <--> f) = 0
                        this_Kerr.x += a*pcdef*2;
                        this_Kerr.y += b*pcdef*2;
                    }
                }
                Kerr[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Kerr;
            }
            break;

        case 1: // compute the SRa tensors, isotropic Raman response
            if (include_Raman && this_SRa_beginning_nonzero > 0) {
                double2 this_Ra;
                this_Ra.x = 0; this_Ra.y = 0; // initialized
                for (int i = this_SRa_beginning_nonzero-1; i < this_SRa_ending_nonzero-1; i++) {
                    midx3 = SRa_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRa_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_At[midx3].x;
                    d = this_At[midx3].y;
                    e = this_At[midx4].x;
                    f = this_At[midx4].y;
            
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f
                            this_Ra.x += SRa[i]*(c*e+d*f);
                        } else {
                            this_Ra.x += SRa[i]*(c*e+d*f);
                            this_Ra.y += SRa[i]*(d*e-c*f);
                        }
                    } else { // (d*e-c*f) + (c <--> e, d <--> f) = 0
                        this_Ra.x += SRa[i]*(c*e+d*f)*2;
                    }
                }
                Ra[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Ra;
            }
            break;

        case 2: // compute the SRb tensors, anisotropic Raman response
            if ( (include_Raman && include_anisoRaman) && this_SRb_beginning_nonzero > 0) {
                double2 this_Rb;
                this_Rb.x = 0; this_Rb.y = 0; // initialized
                for (int i = this_SRb_beginning_nonzero-1; i < this_SRb_ending_nonzero-1; i++) {
                    midx3 = SRb_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRb_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_At[midx3].x;
                    d = this_At[midx3].y;
                    e = this_At[midx4].x;
                    f = this_At[midx4].y;
        
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f
                            this_Rb.x += SRb[i]*(c*e+d*f);
                        } else {
                            this_Rb.x += SRb[i]*(c*e+d*f);
                            this_Rb.y += SRb[i]*(d*e-c*f);
                        }
                    } else { // (d*e-c*f) + (c <--> e, d <--> f) = 0
                        this_Rb.x += SRb[i]*(c*e+d*f)*2;
                    }
                }
                Rb[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Rb;
            }
            break;

        case 3: // compute the spontaneous SRa tensors from the isotropic Raman response
            if (include_Raman && this_SRa_beginning_nonzero > 0) {
                double p, q, r, s; // this_At_noise
                double2 this_Ra_sponRS;
                this_Ra_sponRS.x = 0; this_Ra_sponRS.y = 0; // initialized
                for (int i = this_SRa_beginning_nonzero-1; i < this_SRa_ending_nonzero-1; i++) {
                    midx3 = SRa_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRa_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_At[midx3].x;
                    d = this_At[midx3].y;
                    e = this_At[midx4].x;
                    f = this_At[midx4].y;

                    p = this_At_noise[midx3].x;
                    q = this_At_noise[midx3].y;
                    r = this_At_noise[midx4].x;
                    s = this_At_noise[midx4].y;
            
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) {
                            this_Ra_sponRS.x += SRa[i]*( (p*r+q*s) + (c*r+d*s)*2 );
                        } else {
                            this_Ra_sponRS.x += SRa[i]*( (p*r+q*s) + (c*r+d*s) + (p*e+q*f) );
                            this_Ra_sponRS.y += SRa[i]*( (q*r-p*s) + (d*r-c*s) + (q*e-p*f) );
                        }
                    } else {
                        this_Ra_sponRS.x += SRa[i]*( (p*r+q*s)*2 + (c*r+d*s)*2+(e*p+f*q)*2 );
                    }
                }
                Ra_sponRS[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Ra_sponRS;
            }
            break;

        case 4: // compute the spontaneous SRb tensors from the anisotropic Raman response
            if ( (include_Raman && include_anisoRaman) && this_SRb_beginning_nonzero > 0) {
                double p, q, r, s; // this_At_noise
                double2 this_Rb_sponRS;
                this_Rb_sponRS.x = 0; this_Rb_sponRS.y = 0; // initialized
                for (int i = this_SRb_beginning_nonzero-1; i < this_SRb_ending_nonzero-1; i++) {
                    midx3 = SRb_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRb_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_At[midx3].x;
                    d = this_At[midx3].y;
                    e = this_At[midx4].x;
                    f = this_At[midx4].y;

                    p = this_At_noise[midx3].x;
                    q = this_At_noise[midx3].y;
                    r = this_At_noise[midx4].x;
                    s = this_At_noise[midx4].y;
        
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) {
                            this_Rb_sponRS.x += SRb[i]*( (p*r+q*s) + (c*r+d*s)*2 );
                        } else {
                            this_Rb_sponRS.x += SRb[i]*( (p*r+q*s) + (c*r+d*s) + (p*e+q*f) );
                            this_Rb_sponRS.y += SRb[i]*( (q*r-p*s) + (d*r-c*s) + (q*e-p*f) );
                        }
                    } else {
                        this_Rb_sponRS.x += SRb[i]*( (p*r+q*s)*2 + (c*r+d*s)*2+(e*p+f*q)*2 );
                    }
                }
                Rb_sponRS[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Rb_sponRS;
            }
            break;
    }
}
